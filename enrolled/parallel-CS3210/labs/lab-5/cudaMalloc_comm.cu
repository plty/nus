/**
 * Global Memory (Linear Array)
 * Demonstrates:
 * - Allocation of linear array by host
 * - Passing global memory pointer to device
 * - Method in which host accesses global memory
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
    }

}

__global__ void incrementor(int* numbers)
{
    numbers[threadIdx.x]++;
}

int main(int argc, char **argv)
{
    int *start, *device_mem;
    int i, num_elements;

    hipError_t rc;

    // Ask user for number of elements
    printf("How many elements to increment? ");
    scanf("%d", &num_elements);

    // Seed our RNG
    srand(0);

    // Malloc host memory
    start = (int*)malloc(num_elements * sizeof(int));
    // "Malloc" device memory
    hipMalloc((void **)&device_mem, num_elements * sizeof(int));

    printf("Incrementor input:\n");
    for (i = 0; i < num_elements; i++) {
        start[i] = rand() % 100;
        printf("start[%d] = %d\n", i, start[i]);
    }

    /**
     * Copy a value from start to our CUDA
     */
    rc = hipMemcpy(device_mem, start, num_elements * sizeof(int), hipMemcpyHostToDevice);

    if (rc != hipSuccess)
    {
        printf("Could not copy to device. Reason: %s\n", hipGetErrorString(rc));
    }

    incrementor<<<1, num_elements>>>(device_mem);
    check_cuda_errors();

    // Retrieve data from global memory
    rc = hipMemcpy(start, device_mem, num_elements * sizeof(int), hipMemcpyDeviceToHost);

    if (rc != hipSuccess)
    {
        printf("Could not copy from device. Reason: %s\n", hipGetErrorString(rc));
    }

    printf("Incrementor results:\n");
    for (i = 0; i < num_elements; i++) {
        printf("result[%d] = %d\n", i, start[i]);
    }

    // Free both host and device memory
    free(start);
    hipFree(device_mem);

    return 0;
}
