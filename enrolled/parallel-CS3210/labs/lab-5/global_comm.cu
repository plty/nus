/**
 * Global Memory (Symbol)
 * Demonstrates:
 * - Communication between host and device
 * - Method in which host accesses global memory
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_ELEMENTS 5

__device__ int result[NUM_ELEMENTS];

void check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
    }

}

__global__ void incrementor()
{
    result[threadIdx.x]++;
}

int main(int argc, char **argv)
{
    int start[NUM_ELEMENTS];
    int host_result[NUM_ELEMENTS];
    int i;

    hipError_t rc;

    // Seed our RNG
    srand(0);

    printf("Incrementor input:\n");
    for (i = 0; i < NUM_ELEMENTS; i++) {
        start[i] = rand() % 100;
        printf("start[%d] = %d\n", i, start[i]);
    }

    /**
     * Copy a value from result to host
     */
    rc = hipMemcpyToSymbol(HIP_SYMBOL(result), &start, sizeof(start));

    if (rc != hipSuccess)
    {
        printf("Could not copy to device. Reason: %s\n", hipGetErrorString(rc));
    }

    incrementor<<<1, NUM_ELEMENTS>>>();
    check_cuda_errors();

    // Retrieve data from global memory variable
    rc = hipMemcpyFromSymbol(&host_result, HIP_SYMBOL(result), sizeof(start));

    if (rc != hipSuccess)
    {
        printf("Could not copy from device. Reason: %s\n", hipGetErrorString(rc));
    }

    printf("Incrementor results:\n");
    for (i = 0; i < NUM_ELEMENTS; i++) {
        printf("result[%d] = %d\n", i, host_result[i]);
    }
    return 0;
}
