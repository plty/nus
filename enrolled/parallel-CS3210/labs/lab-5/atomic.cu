/**
 * All threads increments a counter in global memory
 * by one. The difference is that one uses CUDA's atomic function
 * to perform an increment.
 * What to observe/ponder:
 * - What are the values that are printed out?
 * - Are they consistent across runs?
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ __managed__ int counter;

void check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
    }

}

__global__ void non_atomic()
{
    counter++;
}

__global__ void atomic()
{
    atomicAdd(&counter, 1);
}

int main(int argc, char **argv)
{
    // Set up counter
    counter = 0;

    dim3 gridDim(128, 128);
    dim3 blockDim(32, 32);
    non_atomic<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    check_cuda_errors();

    printf("Result from non-atomic increment by 16777216 threads: %d\n", counter);

    // Reset counter
    counter = 0;

    atomic<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    check_cuda_errors();

    printf("Result from atomic increment by 16777216 threads: %d\n", counter);

    return 0;
}