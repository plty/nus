/*
 * Hello World in CUDA
 *
 * CS3210
 *
 * This program start from "hello world" string and should print "HELLO WORLD"
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N       32 

// #define      DISCRETE

__global__ void hello(char *a, int len)
{
        int tid = threadIdx.x;
        if (tid >= len)
                return;
        a[tid] += 'A' - 'a';
}

int main()
{
        // original string
        char a[N] = "hello@world";
        // length
        int len = strlen(a);
        // pointer to the string on device
        char* ad;
        // pointer to the final string on host
        char* ah;
        // CUDA returned error code
        hipError_t rc;


        //allocate space for the string on device (GPU) memory
        hipMalloc((void**)&ad, N);
        hipMemcpy(ad, a, N, hipMemcpyHostToDevice);

        // launch the kernel
        hello<<<1, N>>>(ad, len);
        hipDeviceSynchronize();

	// for discrete GPUs, get the data from device memory to host memory
        hipMemcpy(a, ad, N, hipMemcpyDeviceToHost);
        ah = a;

        // was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

        // print final string
        printf("%s!\n", ah);

        // free memory
        hipFree(ad);

        return 0;
}

