#include "hip/hip_runtime.h"
#include "hash.h"
#include <vector>
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <stdint.h>
#include <cstring>
using namespace std;

__managed__ bool found;
__managed__ unsigned long long ans;
__managed__ unsigned long long target;

// in byte 
__managed__ size_t prefix_length;

long long nativesGetTimeInSeconds() {
    return time(NULL);
}

void check_cuda_errors() {
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess) {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
    }

}

__global__ void solve(uint8_t *prefix_byte_string, unsigned long long init) {
    unsigned long long idx = init + threadIdx.x;
    uint8_t* byte_string;

    // Append idx to byte_string
    size_t sz = (prefix_length + sizeof(long long)) * sizeof(uint8_t);
    byte_string = (uint8_t*) malloc(sz);
    memcpy(byte_string, prefix_byte_string ,prefix_length);

    uint8_t* suffix = byte_string + prefix_length;
    for(int i = 0 ; i < 8 ; i ++) {
        suffix[7 - i] = idx >> (i * 8);
    }

    // Compute hash
    uint8_t result[32];
    sha256(result, byte_string, prefix_length + sizeof(long long));

    // Check for result
    unsigned long long prefix_value = 0;
    for(int i = 0 ; i < 8 ; i ++) {
        prefix_value <<= 8;
        prefix_value |= result[i];
    }

    if(prefix_value < target) {
        found = true;
        atomic_exch(&ans, idx);
    }

    free(byte_string);
}

int main() {
    unsigned long long cur_time = nativesGetTimeInSeconds();
    vector<uint8_t> byte_string;

    // Add time;
    vector<uint8_t> tmp;
    for(int i = 0 ; i < 64 ; i += 8) {
        tmp.push_back(cur_time);
        cur_time >>= 8;
    }
    reverse(tmp.begin(), tmp.end());
    byte_string.insert(byte_string.end(), tmp.begin(), tmp.end());

    string digest; cin >> digest;

    // Add previous digest
    for (int i = 0; i < digest.size() ; i += 2) {
        uint8_t p;
        sscanf(digest.c_str() + i, "%2x", &p);
        byte_string.push_back(p);
    }

    const string nusnet_id = "gganteng";

    // Add previous digest
    for (int i = 0; i < nusnet_id.size(); i += 2) {
        uint8_t p;
        sscanf(nusnet_id.c_str() + i, "%2x", &p);
        byte_string.push_back(p);
    }


    // Get target
    cin >> target;


    check_cuda_errors();

    // Copy precomputed prefix to device
    uint8_t* prefix_byte_string;
    prefix_length = byte_string.size();
    hipMalloc((void**)&prefix_byte_string, prefix_length);

    check_cuda_errors();


    uint8_t x[byte_string.size()];
    for (int i = 0; i < byte_string.size(); i ++) {
        x[i] = byte_string[i];
    }
    hipMemcpy(prefix_byte_string, x, prefix_length, hipMemcpyHostToDevice );

    long long blockSize = 64;

    // coba coba
    found = false;

    for(long long  i = 0; i <= ULLONG_MAX ; i += blockSize) {
        // cout << "Trying with anchor: " << i << endl;
        solve<<<1, blockSize>>>(prefix_byte_string, i);
        hipDeviceSynchronize();
        check_cuda_errors();
        if(found) {
            cout << "KETEMU COI " << ans << endl;
            break;
        }
    }

    return 0;
}
